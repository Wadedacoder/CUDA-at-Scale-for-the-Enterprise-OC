#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <string.h>
#include <fstream>
#include <iostream>
#include "stbi_image.h"
#include "stbi_image_write.h"
#include <hip/hip_runtime.h>
#include <npp.h>

// #include <hip/hip_runtime_api.h>
// #include <helper_string.h>

bool printfNPPinfo(int argc, char *argv[])
{
  const NppLibraryVersion *libVer = nppGetLibVersion();

  printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,
         libVer->build);

  int driverVersion, runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
         (driverVersion % 100) / 10);
  printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
         (runtimeVersion % 100) / 10);

  // Min spec is SM 1.0 devices
  // bool bVal = checkCudaCapabilities(1, 0);
  // return bVal;
  return true;
}

int main(int argc, char *argv[])
{
  printf("%s Starting...\n\n", argv[0]);

  // try
  {
    std::string sFilename;
    char *filePath;

    // findCudaDevice(argc, (const char **)argv);

    if (printfNPPinfo(argc, argv) == false)
    {
      exit(EXIT_SUCCESS);
    }

    // We need to load the image from disk
    if (argc == 1)
    {
      // No arguments were passed
      printf("No image data passed\n");
      exit(EXIT_FAILURE);
    }
    else
    {
      // Load the image from disk
      sFilename = std::string(argv[1]);
      filePath = argv[1];
    }

    int width, height, channels;
    unsigned char *img = stbi_load(filePath, &width, &height, &channels, 0);

    // Check if the image was loaded
    if (img == NULL)
    {
      printf("Error loading image\n");
      exit(EXIT_FAILURE);
    }

    // Convert the image to NPP format
    Npp8u *d_src = NULL;
    NppiSize oSizeROI;
    oSizeROI.width = width;
    oSizeROI.height = height;

    hipMalloc((void **)&d_src, width * height * channels * sizeof(Npp8u));
    hipMemcpy(d_src, img, width * height * channels * sizeof(Npp8u),
               hipMemcpyHostToDevice);

    // Convert the image to grayscale
    Npp8u *d_dst = NULL;
    hipMalloc((void **)&d_dst, width * height * sizeof(Npp8u));

    NppStatus npp_status;
    npp_status = nppiRGBToGray_8u_C3C1R(d_src, width * channels, d_dst, width,
                                         oSizeROI);

    if (npp_status != NPP_SUCCESS)
    {
      printf("Error converting image to grayscale\n");
      exit(EXIT_FAILURE);
    }

    // Do sobel edge detection
    /*NppStatus nppiFilterSobelHoriz_8u_C1R(const Npp8u *pSrc, Npp32s nSrcStep, Npp8u *pDst, Npp32s nDstStep, NppiSize oSizeROI)*/
    Npp8u *d_sobel = NULL;
    hipMalloc((void **)&d_sobel, width * height * sizeof(Npp8u));

    npp_status = nppiFilterSobelHoriz_8u_C1R(d_dst, width, d_sobel, width,
                                              oSizeROI);
    
    if (npp_status != NPP_SUCCESS)
    {
      printf("Error doing sobel edge detection\n");
      exit(EXIT_FAILURE);
    }


    // Copy the image back to the host
    unsigned char *sobel = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    hipMemcpy(sobel, d_sobel, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save the image to disk
    stbi_write_png("sobel.png", width, height, 1, sobel, width);

    // Free the memory
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_sobel);
    free(sobel);

    // Free the image
    stbi_image_free(img);

    return;
  }
}
